#include "hip/hip_runtime.h"


#ifndef _STRING_BITONICSORT_KERNEL_
#define _STRING_BITONICSORT_KERNEL_

#include <stdio.h>
#include <common.cu>
#include <StringCmp.cu>

#define NUM_BLOCKS_CHUNK (512)
#define	STRING_NUM_THREADS_CHUNK (256)
#define CHUNK_SIZE (NUM_BLOCKS_CHUNK*STRING_NUM_THREADS_CHUNK)
#define NUM_CHUNKS_R (NUM_RECORDS_R/CHUNK_SIZE)






__global__ void
string_partBitonicSortKernel( void* d_rawData, int totalLenInBytes,cmp_type_t* d_R, unsigned int numRecords, int chunkIdx, int unitSize)
{
	__shared__ cmp_type_t shared[STRING_NUM_THREADS_CHUNK];

	int tx = threadIdx.x;
	int bx = blockIdx.x;

	//load the data
	int dataIdx = chunkIdx*CHUNK_SIZE+bx*blockDim.x+tx;
	int unitIdx = ((NUM_BLOCKS_CHUNK*chunkIdx + bx)/unitSize)&1;
	shared[tx] = d_R[dataIdx];
	__syncthreads();
	int ixj=0;
	int a=0;
	cmp_type_t temp1;
	cmp_type_t temp2;
	int k = STRING_NUM_THREADS_CHUNK;

	if(unitIdx == 0)
	{
		for (int j = (k>>1); j>0; j =(j>>1))
		{
			ixj = tx ^ j;
			//a = (shared[tx].y - shared[ixj].y);				
			temp1=shared[tx];
			temp2= shared[ixj];
			if (ixj > tx) {
				//a=temp1.y-temp2.y;
				//a=compareString((void*)(((char4*)d_rawData)+temp1.x),(void*)(((char4*)d_rawData)+temp2.x)); 
				a=getCompareValue(d_rawData, temp1.x, temp2.x);
				if ((tx & k) == 0) {
					if ( (a>0)) {
						shared[tx]=temp2;
						shared[ixj]=temp1;
					}
				}
				else {
					if ( (a<0)) {
						shared[tx]=temp2;
						shared[ixj]=temp1;
					}
				}
			}
				
			__syncthreads();
		}
	}
	else
	{
		for (int j = (k>>1); j>0; j =(j>>1))
		{
			ixj = tx ^ j;
			temp1=shared[tx];
			temp2= shared[ixj];
			
			if (ixj > tx) {					
				//a=temp1.y-temp2.y;					
				//a=compareString((void*)(((char4*)d_rawData)+temp1.x),(void*)(((char4*)d_rawData)+temp2.x));
				a=getCompareValue(d_rawData, temp1.x, temp2.x);
				if ((tx & k) == 0) {
					if( (a<0))
					{
						shared[tx]=temp2;
						shared[ixj]=temp1;
					}
				}
				else {
					if( (a>0))
					{
						shared[tx]=temp2;
						shared[ixj]=temp1;
					}
				}
			}
			
			__syncthreads();
		}
	}

	d_R[dataIdx] = shared[tx];
}

__global__ void
string_unitBitonicSortKernel(void* d_rawData, int totalLenInBytes, cmp_type_t* d_R, unsigned int numRecords, int chunkIdx )
{
	__shared__ cmp_type_t shared[STRING_NUM_THREADS_CHUNK];

	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int unitIdx = (NUM_BLOCKS_CHUNK*chunkIdx + bx)&1;

	//load the data
	int dataIdx = chunkIdx*CHUNK_SIZE+bx*blockDim.x+tx;
	shared[tx] = d_R[dataIdx];
	__syncthreads();

	cmp_type_t temp1;
	cmp_type_t temp2;
	int ixj=0;
	int a=0;
	if(unitIdx == 0)
	{
		for (int k = 2; k <= STRING_NUM_THREADS_CHUNK; (k =k<<1))
		{
			// bitonic merge:
			for (int j = (k>>1); j>0; (j=j>>1))
			{
				ixj = tx ^ j;	
				temp1=shared[tx];
				temp2= shared[ixj];
				if (ixj > tx) {					
					//a=temp1.y-temp2.y;
					//a=compareString((void*)(((char4*)d_rawData)+temp1.x),(void*)(((char4*)d_rawData)+temp2.x));
					a=getCompareValue(d_rawData, temp1.x, temp2.x);
					if ((tx & k) == 0) {
						if ( (a>0)) {
							shared[tx]=temp2;
							shared[ixj]=temp1;
						}
					}
					else {
						if ( (a<0)) {
							shared[tx]=temp2;
							shared[ixj]=temp1;
						}
					}
				}
				
				__syncthreads();
			}
		}
	}
	else
	{
		for (int k = 2; k <= STRING_NUM_THREADS_CHUNK; (k =k<<1))
		{
			// bitonic merge:
			for (int j = (k>>1); j>0; (j=j>>1))
			{
				ixj = tx ^ j;
				temp1=shared[tx];
				temp2= shared[ixj];
				if (ixj > tx) {					
					//a=temp1.y-temp2.y;
					//a=compareString((void*)(((char4*)d_rawData)+temp1.x),(void*)(((char4*)d_rawData)+temp2.x));
					a=getCompareValue(d_rawData, temp1.x, temp2.x);
					if ((tx & k) == 0) {
						if( (a<0))
						{
							shared[tx]=temp2;
							shared[ixj]=temp1;
						}
					}
					else {
						if( (a>0))
						{
							shared[tx]=temp2;
							shared[ixj]=temp1;
						}
					}
				}
				
				__syncthreads();
			}
		}

	}

	d_R[dataIdx] = shared[tx];
}

__global__ void
string_bitonicKernel( void* d_rawData, int totalLenInBytes, cmp_type_t* d_R, unsigned int numRecords, int k, int j)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tid = threadIdx.x;
	int dataIdx = by*gridDim.x*blockDim.x + bx*blockDim.x + tid;

	int ixj = dataIdx^j;

	if( ixj > dataIdx )
	{
		cmp_type_t tmpR = d_R[dataIdx];
		cmp_type_t tmpIxj = d_R[ixj];
		if( (dataIdx&k) == 0 )
		{
			//if( tmpR.y > tmpIxj.y )
			//if(compareString((void*)(((char4*)d_rawData)+tmpR.x),(void*)(((char4*)d_rawData)+tmpIxj.x))==1) 
			if(getCompareValue(d_rawData, tmpR.x, tmpIxj.x)==1)
			{
				d_R[dataIdx] = tmpIxj;
				d_R[ixj] = tmpR;
			}
		}
		else
		{
			//if( tmpR.y < tmpIxj.y )
			//if(compareString((void*)(((char4*)d_rawData)+tmpR.x),(void*)(((char4*)d_rawData)+tmpIxj.x))==-1) 
			if(getCompareValue(d_rawData, tmpR.x, tmpIxj.x)==-1)
			{
				d_R[dataIdx] = tmpIxj;
				d_R[ixj] = tmpR;
			}
		}
	}
}


#endif // #ifndef _TEMPLATE_KERNEL_H_
