#include "hip/hip_runtime.h"
#ifndef STRING_BITONIC_PROC_H
#define STRING_BITONIC_PROC_H
#include "string_bitonicProc_kernel.cu"
//#define NUM_BLOCK_PER_CHUNK_BITONIC_SORT 512//b256

/*
@totalLenInBytes, is not used. 
*/
void string_bitonicSortMultipleBlocks(void* d_rawData, int totalLenInBytes, cmp_type_t * d_values, int* d_bound, int numBlock, cmp_type_t * d_output)
{
	int numThreadsPerBlock_x=SHARED_MEM_INT2;
	int numThreadsPerBlock_y=1;
	int numBlock_x=NUM_BLOCK_PER_CHUNK_BITONIC_SORT;
	int numBlock_y=1;
	int numChunk=numBlock/numBlock_x;
	if(numBlock%numBlock_x!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*numBlock_x;
		end=start+numBlock_x;
		if(end>numBlock)
			end=numBlock;
		//printf("bitonicSortMultipleBlocks_kernel: %d, range, %d, %d\n", i, start, end);
		string_bitonicSortMultipleBlocks_kernel<<<grid,thread>>>(d_rawData, totalLenInBytes, d_values, d_bound, start, end-start, d_output);
		hipDeviceSynchronize();
	}
//	hipDeviceSynchronize();
}

void initialize(cmp_type_t *d_data, int rLen, cmp_type_t value)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		initialize_kernel<<<grid,thread>>>(d_data, start, rLen, value);
	} 
	hipDeviceSynchronize();
}

void int4toint2(int4 *d_data, int rLen, Record* d_output)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		int4toint2_kernel<<<grid,thread>>>(d_data, start, rLen, d_output);
	} 
	hipDeviceSynchronize();
}


void getIntYArray(Record *d_data, int rLen, int* d_output)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		getIntYArray_kernel<<<grid,thread>>>(d_data, start, rLen, d_output);
	} 
	hipDeviceSynchronize();
}

void getXYArray(cmp_type_t *d_data, int rLen, Record* d_output)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		getXYArray_kernel<<<grid,thread>>>(d_data, start, rLen, d_output);
	} 
	hipDeviceSynchronize();
}

void getZWArray(cmp_type_t *d_data, int rLen, Record* d_output)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		getZWArray_kernel<<<grid,thread>>>(d_data, start, rLen, d_output);
	} 
	hipDeviceSynchronize();
}



void setXYArray(cmp_type_t *d_data, int rLen, Record* d_value)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		setXYArray_kernel<<<grid,thread>>>(d_data, start, rLen, d_value);
	} 
	hipDeviceSynchronize();
}

void setZWArray(cmp_type_t *d_data, int rLen, Record* d_value)
{
	int numThreadsPerBlock_x=512;
	int numThreadsPerBlock_y=1;
	int numBlock_x=512;
	int numBlock_y=1;
	int chunkSize=numBlock_x*numThreadsPerBlock_x;
	int numChunk=rLen/chunkSize;
	if(rLen%chunkSize!=0)
		numChunk++;

	dim3  thread( numThreadsPerBlock_x, numThreadsPerBlock_y, 1);
	dim3  grid( numBlock_x, numBlock_y , 1);
	int i=0;
	int start=0;
	int end=0;
	for(i=0;i<numChunk;i++)
	{
		start=i*chunkSize;
		end=start+chunkSize;
		if(end>rLen)
			end=rLen;
		setZWArray_kernel<<<grid,thread>>>(d_data, start, rLen, d_value);
	} 
	hipDeviceSynchronize();
}

#endif
