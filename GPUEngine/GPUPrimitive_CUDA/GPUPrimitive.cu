#include "hip/hip_runtime.h"


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <TestAll.cu>
#include "TestJoin.cu"
#include "GPUDB_AccessMethod.cu"
#include "GPUDB_Operator.cu"



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
	//int numPart=32;
	//printf("%d",1<<(log2((int)(SHARED_MEMORY_PER_PROCESSOR/(numPart*sizeof(int))))));
	//printf("%d, %d", '\0', 'A'); 
   
	testAllPrimitive(argc, argv);
	//testAllJoin(argc, argv);
	//test_Operators(argc,argv);
	//test_AccessMethods(argc,argv);
	
	//testINLJ(1024*1024*16, 1024*1024*16);

	//testHashSearch(1024*1024, 1024*1024);
	//testTreeSearch(1024*1024, 1024*1024);

	//testHJ(1024*1024*16, 1024*1024*16);	
	//testRadixSort(1024*1024*8);

	//testINLJ( 1024*1024*16, 1024*1024*16 );
	//testNINLJ( 1024*1024, 1024*1024);
	//testSMJ(1024*1024*16, 1024*1024*16);
	/*int rLen = 1024*1024*8;
	int numThread = 256;
	int numBlock = 256;
	testAggAfterGroupByImpl( rLen, REDUCE_AVERAGE, numThread, numBlock );*/
	

	//testProjection( 1024*1024*16, 1024*1024*16*0.01, 128, 128 );

	/*int rLen = 1024*1024;
	int numPart = 8;
	testPartition(rLen, numPart );*/

	//testRadixSort( 1024*1024*1 );

	//testFilterImpl( 1024*1024*16 );

	//testGroupByImpl( 1024*1024*16 );

	//testReduceImpl( 1024*1024*16, REDUCE_MAX);

	//testMapImpl( 1024*1024*16, 512, 1024);

	//testGather( 1024*1024*16 );
	//testScatter( 1024*1024*16 );

	//testSplit(1024*1024*16, 64, 64, 1024);

	//testSelection( 1024*1024*16 );

	//testScan( 1024*1024*16 );

	//testQSort( 1024*1024*16 );

	return 0;
}


